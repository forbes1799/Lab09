
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void helloKernel(int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N) {
    printf("thread %d of block %d (dim: %d): iter %d of %d\n", threadIdx.x, blockIdx.x, blockDim.x, i, N);
  }
}


int main(void) {
  int N=10,numGPUs;

  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  if (numGPUs >= 1 ) {

  printf("hello on CPU\n");

  /* call GPU kernel  */
  helloKernel<<<4,3>>> (N);

  printf("hello again from CPU\n");

}

  else {
   printf("no GPU present\n");
}

}
