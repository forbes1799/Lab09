
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void vectorAdd(float *z, float *x, float *y, float A, int num){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	while(i < num){
		z[i] = A * x[i] + y[i];
		i += blockDim.x * gridDim.x;
	}
}

int main(void){
	//declaration of host variables
	const int num = 1000000;
	float *z, *x, *y;

	float *z_dev, *x_dev, *y_dev;

	float A = 34;
	
	//initialising host variables
	z = (float*) malloc(num * sizeof(float));
	x = (float*) malloc(num * sizeof(float));
	y = (float*) malloc(num * sizeof(float));

	hipMalloc(&x_dev, num*sizeof(float));
	hipMalloc(&y_dev, num*sizeof(float));
	hipMalloc(&z_dev, num*sizeof(float));


	for(int i = 0; i < num; i++){
		x[i] = i;
		y[i] = 7 * i;
	}

	hipMemcpy(x_dev, x, num*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_dev, y, num*sizeof(float), hipMemcpyHostToDevice);

	vectorAdd<<<8, 128>>>(z_dev, x_dev, y_dev, A, num);

	hipMemcpy(z, z_dev, num * sizeof(float), hipMemcpyDeviceToHost);
	
	//postprocessing: output to terminal
	for(int j = 0; j < num; j++){
		printf("%f ", z[j]);
	}
	
	return 0;
}
